
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stack>
#include <stdlib.h>
#include <time.h>

class MazeCell
{
public:
    int coordX = 0;
    int coordY = 0;
    bool partOfPath = false;
    std::vector<int> walls{0, 1, 2, 3};
    bool visit = false;
};

class Maze
{
public:
    void createMazeGrid(int rows, int columns, int randomSeed);
    std::vector<MazeCell> visitedCells;
    int amountOfVisitedCells = 0;
    std::vector<std::vector<MazeCell>> createMaze(std::vector<std::vector<MazeCell>> mazeGrid, int coordX, int coordY, std::vector<MazeCell> visitedCells, int amountOfVisitedCells);
    void printMaze(std::vector<std::vector<MazeCell>> mazeGrid);
};

void Maze::createMazeGrid(int rows, int columns, int randomSeed)
{
    if (randomSeed != 0)
        srand(randomSeed);

    std::string partOfPath = " ";
    MazeCell singleCell;
    std::vector<std::vector<MazeCell>> mazeGrid;
    for (int i = 0; i < rows; i++)
    {
        std::vector<MazeCell> mazeColumn;
        for (int j = 0; j < columns; j++)
        {
            mazeColumn.push_back(singleCell);
        }
        mazeGrid.push_back(mazeColumn);
    }
    Maze newMaze;
    MazeCell startingMazeCell;
    startingMazeCell.visit = true;
    newMaze.visitedCells.push_back(startingMazeCell);
    mazeGrid = newMaze.createMaze(mazeGrid, startingMazeCell.coordX, startingMazeCell.coordY, newMaze.visitedCells, newMaze.amountOfVisitedCells);
    newMaze.printMaze(mazeGrid);
}

int removePossibleWall(std::vector<std::vector<MazeCell>> mazeGrid, std::vector<int> possibleWalls, int coordX, int coordY)
{
    if (possibleWalls.size() > 0)
    {

        int removedWallIndex = rand() % possibleWalls.size();
        int removedWall = possibleWalls.at(removedWallIndex);
        switch (removedWall)
        {
        case 0: // wall 0 is the left wall
            if (mazeGrid.at(coordX - 1).at(coordY).visit == true)
            {
                possibleWalls.erase(possibleWalls.begin() + removedWallIndex);
                return removePossibleWall(mazeGrid, possibleWalls, coordX, coordY);
            }
            else
            {
                return removedWall;
            }
            break;
        case 1: // wall 1 is the back wall
            if (mazeGrid.at(coordX).at(coordY - 1).visit == true)
            {
                possibleWalls.erase(possibleWalls.begin() + removedWallIndex);
                return removePossibleWall(mazeGrid, possibleWalls, coordX, coordY);
            }
            else
            {
                return removedWall;
            }
            break;
        case 2: // wall 2 is the right wall
            if (mazeGrid.at(coordX + 1).at(coordY).visit == true)
            {
                possibleWalls.erase(possibleWalls.begin() + removedWallIndex);
                return removePossibleWall(mazeGrid, possibleWalls, coordX, coordY);
            }
            else
            {
                return removedWall;
            }
            break;
        case 3: // wall 3 is the bottom wall
            if (mazeGrid.at(coordX).at(coordY + 1).visit == true)
            {
                possibleWalls.erase(possibleWalls.begin() + removedWallIndex);
                return removePossibleWall(mazeGrid, possibleWalls, coordX, coordY);
            }
            else
            {
                return removedWall;
            }
            break;
        }
    }
    return -1; //if no walls (adjacent tiles) are available, return -1 to let the createMaze function know
}

std::vector<std::vector<MazeCell>> Maze::createMaze(std::vector<std::vector<MazeCell>> mazeGrid, int coordX, int coordY, std::vector<MazeCell> visitedCells, int amountOfVisitedCells)
{
    if (amountOfVisitedCells >= mazeGrid.size() * mazeGrid.at(0).size())
    {
        return mazeGrid;
    }
    Maze newMaze;
    MazeCell currentMazeCell = mazeGrid.at(coordX).at(coordY);
    currentMazeCell.coordX = coordX; // currentMazeCell coordinates are 0, 0, this line changes that
    currentMazeCell.coordY = coordY;
    visitedCells.push_back(currentMazeCell);
    if (currentMazeCell.visit == false)
    {
        if (coordX == mazeGrid.size() - 1 && coordY == mazeGrid.at(0).size() - 1)
        {
            for (int i = 0; i < visitedCells.size(); i++)
            {
                int x = visitedCells.at(i).coordX;
                int y = visitedCells.at(i).coordY;
                mazeGrid.at(x).at(y).partOfPath = true;
            }
            currentMazeCell.partOfPath = true;
        }
        currentMazeCell.visit = true;
        amountOfVisitedCells = amountOfVisitedCells + 1;
    }
    // create a list of possible walls to break
    std::vector<int> possibleWalls;
    if (coordX > 0)
        possibleWalls.push_back(0);
    if (coordX < mazeGrid.size() - 1)
        possibleWalls.push_back(2);
    if (coordY > 0)
        possibleWalls.push_back(1);
    if (coordY < mazeGrid.at(0).size() - 1)
        possibleWalls.push_back(3);
    int removedWall = removePossibleWall(mazeGrid, possibleWalls, coordX, coordY);
    if (removedWall == -1)
    {
        while (visitedCells.back().coordX == currentMazeCell.coordX && visitedCells.back().coordY == currentMazeCell.coordY)
        {
            visitedCells.pop_back();
        }
        mazeGrid.at(coordX).at(coordY) = currentMazeCell;
        return newMaze.createMaze(mazeGrid, visitedCells.back().coordX, visitedCells.back().coordY, visitedCells, amountOfVisitedCells);
    }
    for (int i = 0; i < currentMazeCell.walls.size(); i++)
    {
        if (currentMazeCell.walls.at(i) == removedWall)
        {
            currentMazeCell.walls.erase(currentMazeCell.walls.begin() + i);
            break;
        }
    }
    mazeGrid.at(coordX).at(coordY) = currentMazeCell;
    switch (removedWall)
    {
    case 0:
        return newMaze.createMaze(mazeGrid, coordX - 1, coordY, visitedCells, amountOfVisitedCells);
        coordX = coordX - 1;
    case 1:
        coordY = coordY - 1;
        return newMaze.createMaze(mazeGrid, coordX, coordY, visitedCells, amountOfVisitedCells);
    case 2:
        coordX++;
        return newMaze.createMaze(mazeGrid, coordX, coordY, visitedCells, amountOfVisitedCells);
    case 3:
        coordY++;
        return newMaze.createMaze(mazeGrid, coordX, coordY, visitedCells, amountOfVisitedCells);
    }
    return mazeGrid; // never reached
}

std::string checkForWall(std::vector<std::vector<MazeCell>> mazeGrid, int coordY, int coordX, int oddOrEvenRow)
{
    if (oddOrEvenRow == 1)
    {
        for (int i = 0; i < mazeGrid.at(coordX).at(coordY).walls.size(); i++)
        {
            if (mazeGrid.at(coordX).at(coordY).walls.at(i) == 1)
            {
                try
                {
                    for (int i = 0; i < mazeGrid.at(coordX).at(coordY - 1).walls.size(); i++)
                    {
                        if (mazeGrid.at(coordX).at(coordY - 1).walls.at(i) == 3)
                        {
                            return "+---";
                        }
                    }
                    return "+   ";
                }
                catch (std::out_of_range)
                {
                    return "+---";
                }
            }
        }
        return "+   ";
    }
    else
    {
        for (int i = 0; i < mazeGrid.at(coordX).at(coordY).walls.size(); i++)
        {
            if (mazeGrid.at(coordX).at(coordY).walls.at(i) == 0)
            {
                try
                {
                    for (int i = 0; i < mazeGrid.at(coordX - 1).at(coordY).walls.size(); i++)
                    {
                        if (mazeGrid.at(coordX - 1).at(coordY).walls.at(i) == 2)
                        {
                            if (mazeGrid.at(coordX).at(coordY).partOfPath)
                            {
                                return "| . ";
                            }
                            else
                            {
                                return "|   ";
                            }
                        }
                    }
                }
                catch (std::out_of_range)
                {
                    if (mazeGrid.at(coordX).at(coordY).partOfPath)
                    {
                        return "| . ";
                    }
                    else
                    {
                        return "|   ";
                    }
                }
            }
        }
        if (mazeGrid.at(coordX).at(coordY).partOfPath)
        {
            return "  . ";
        }
        else
        {
            return "    ";
        }
        return ""; //never reached
    }
}

void Maze::printMaze(std::vector<std::vector<MazeCell>> mazeGrid)
{
    int columns = mazeGrid.size();
    int rows = mazeGrid.at(0).size() * 2 + 1;
    for (int i = 0; i < rows - 1; i++)
    {
        if (i % 2)
        { //for the rows that have the horizontal walls |
            for (int j = 0; j < columns; j++)
            {
                std::cout << checkForWall(mazeGrid, (i - 1) / 2, j, 0);
            }
            std::cout << "|\n";
        }
        else //for the rows that have the vertical walls +---
        {
            for (int j = 0; j < columns; j++)
            {
                std::cout << checkForWall(mazeGrid, i / 2, j, 1);
            }
            std::cout << "+\n";
        }
    }
    for (int i = 0; i < columns; i++)
    {
        std::cout << "+---";
    }
    std::cout << "+\n";
}

int main(int argc, char** argv)
{
    Maze newMaze;
    int columns = 0;
    int rows = 0;
    int randomSeed = time(0);
    const clock_t begin_time = clock(); 
    if (argc >= 2) 
    {
        rows = atoi(argv[1]);
    }
    if (argc >= 3) 
    {
        columns = atoi(argv[2]);
    }
    if (argc >= 4) 
    {
        randomSeed = atoi(argv[3]);
    }
    if (columns * rows == 0)
    {
        std::cout << "Error: Not enough arguments provided\n";
        return - 1;
    }
    newMaze.createMazeGrid(columns, rows, randomSeed);
    std::cout << clock () - begin_time << std::endl;
    return 0;
}